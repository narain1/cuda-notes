#include <stdio.h>
#include <hip/hip_runtime.h>

#define DEBUG

__global__ void sgemm(float *a, float *b, float *c, int n_ar, int n_ac,
                            int n_br, int n_bc) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n_ar || col >= n_bc) return;

    float sum = 0;
    for (int k=0; k<n_ac; k++) {  // everything is being fetched from global mem
        sum += a[row * n_ac + k] * b[k * n_bc + col];
    }

    c[row * n_bc + col] = sum;
}

inline unsigned int cdiv(unsigned int a, unsigned int b) {
    return (a + b - 1)/b;
}

extern "C" {
    void cuda_gemm(float *a, float *b, float *c, int aw, int ah, int bw, int bh) {
        float *d_a, *d_b, *d_c;
        
        hipMalloc((void **)&d_a, aw * ah * sizeof(float));
        hipMalloc((void **)&d_b, bw * bh * sizeof(float));
        hipMalloc((void **)&d_c, aw * bh * sizeof(float));

        hipMemcpy(d_a, a, aw * ah * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, bw * bh * sizeof(float), hipMemcpyHostToDevice);

        dim3 blockDim(16, 16);
        dim3 gridDim(cdiv(bw, blockDim.x), cdiv(ah, blockDim.y));

        sgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c,
                                        ah, aw,
                                        bw, bh);

        hipError_t cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "mm kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }

        hipMemcpy(c, d_c, aw * bh *sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

    }
}
